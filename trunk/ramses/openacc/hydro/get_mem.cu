#include <hip/hip_runtime.h> // to get memory on the device
 // to get device count

extern "C" 
{
void get_dev_mem(size_t& total, size_t& free) 
{
	hipMemGetInfo(&free, &total);
}
}